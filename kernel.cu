#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <>


__global__ void cellsKernel(int *cells, int height, int width, int *resultCells,
			    int *borderTop, int *borderRight, int *bordertBot, int *borderLeft)
{
	int worldSize = height * width;
	int currentCellX, currentCellY, aliveCells, currentRow;

	int N, NE, E, SE, S, SW, W, NW;

	for (int cellId = blockIdx.x * blockDim.x + threadIdx.x; cellId < worldSize; cellId += blockDim.x * gridDim.x) {
		currentCellY = cellId % width; // cell's index from the **matrix**
		currentCellX = cellId - currentCellY; // the number of cells in the **matrix** until the current one
		currentRow = cellId / width;

		aliveCells = 0;

		N = (currentRow == 0) ? borderTop[currentCellY] : cells[currentCellX - width + currentCellY];
		S = (currentRow + 1 == height) ? bordertBot[currentCellY] : cells[currentCellX + width + currentCellY];
		W = (currentCellY == 0) ? borderLeft[currentRow + 1] : cells[currentCellX + currentCellY - 1];
		E = (currentCellY + 1 == width) ? borderRight[currentRow] : cells[currentCellX + currentCellY + 1];

		if (currentRow == 0)
			NE = borderTop[currentCellY + 1];
		else if (currentCellY + 1 == width)
			NE = borderRight[currentRow - 1];
		else
			NE = cells[currentCellX - width + currentCellY + 1];
		if (currentCellY == 0)
			NW = borderLeft[currentCellY];
		else if (currentRow == 0)
			NW = borderTop[currentCellY - 1];		
		else
			NW = cells[currentCellX - width + currentCellY - 1];
		if (currentRow + 1 == height)
			SE = bordertBot[currentCellY + 1];
		else if (currentCellY + 1 == width)
			SE = borderRight[currentRow + 1];
		else
			SE = cells[currentCellX + width + currentCellY + 1];
		if (currentCellY == 0)
			SW = borderLeft[currentRow + 2];
		else if (currentRow + 1 == height)
			SW = bordertBot[currentCellY - 1];
		else
			SW = cells[currentCellX + width + currentCellY - 1];

		aliveCells = N + S + E + W + NE + SE + SW + NW;

		resultCells[currentCellX + currentCellY] = (aliveCells == 3 || (aliveCells == 2 && cells[currentCellX + currentCellY] == 1)) ? 1 : 0;
	}
}

void computeCells(int *&cells, int height, int width, int *&resultCells, int threadsCount,
		  int *borderTop, int *borderRight, int *borderBot, int *borderLeft)
{
	if ((width * height) % threadsCount != 0) {
		fprintf(stderr, "%s", "The product of square dimensions must be multiple of the number of threads!\n");
		printf("%s", "The product of square dimensions must be multiple of the number of threads!\n");
		exit(1);
	}

	int blocksCount = min(32768, (height * width) / threadsCount);
	
	cellsKernel <<<blocksCount, threadsCount >>> (cells, height, width, resultCells, borderTop, borderRight, borderBot, borderLeft);
}

int getGreatestDivisor(int n)
{
	int res = n;

	for (int i = 2; i <= sqrt(n); i++) {
		while (res % i == 0) {
			if (res <= 1024)
				return res;
			res /= i;
		}
		if (res <= 1024)
			return res;
	}
	
	return res;
}

extern "C"  int* newGeneration(int *h_cells, int *h_borderTop, int *h_borderBot,
		   int *h_borderRight, int *h_borderLeft, int height, int width)
{
	int *d_cells, *d_resultCells, *d_borderTop, *d_borderRight, *d_borderBot, *d_borderLeft;

	int worldSize = height * width;
	int num_threads = height * width;

	hipMalloc(&d_cells, worldSize * sizeof(int));

	hipMalloc(&d_resultCells, worldSize * sizeof(int));

	hipMalloc(&d_borderTop, (width + 1) * sizeof(int));

	hipMalloc(&d_borderRight, (height) * sizeof(int));

	hipMalloc(&d_borderBot, (width + 1) * sizeof(int));

	hipMalloc(&d_borderLeft, (height + 2) * sizeof(int));

	hipMemcpy(d_cells, h_cells, worldSize * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_borderTop, h_borderTop, width + 1, hipMemcpyHostToDevice);

	hipMemcpy(d_borderBot, h_borderBot, width + 1, hipMemcpyHostToDevice);

	hipMemcpy(d_borderRight, h_borderRight, height, hipMemcpyHostToDevice);

	hipMemcpy(d_borderLeft, h_borderLeft, height + 2, hipMemcpyHostToDevice);

	num_threads = min(getGreatestDivisor(num_threads), width);

	computeCells(d_cells, height, width, d_resultCells, num_threads, d_borderTop,
		     d_borderRight, d_borderBot, d_borderLeft);

	hipMemcpy(h_cells, d_resultCells, worldSize * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_cells);
	hipFree(d_resultCells);
	hipFree(d_borderBot);
	hipFree(d_borderLeft);
	hipFree(d_borderRight);
	hipFree(d_borderTop);

	return h_cells;
}

